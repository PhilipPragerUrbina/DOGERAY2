﻿//class for loaading 3d files
#include "loader.hpp"
#include "window.hpp"
#include "gui.hpp"
#include "tracekernel.hpp"
#include "bvhtree.hpp"
int main(int argc, char* args[])
{
    //display title
    //The R is for correct multiline formatting
    std::cout << R"(
 ______   ______   _______    ______   ______    ________   __  __     _____       
/_____/\ /_____/\ /______/\  /_____/\ /_____/\  /_______/\ /_/\/_/\   /_____/\     
\:::_ \ \\:::_ \ \\::::__\/__\::::_\/_\:::_ \ \ \::: _  \ \\ \ \ \ \  \:::_:\ \    
 \:\ \ \ \\:\ \ \ \\:\ /____/\\:\/___/\\:(_) ) )_\::(_)  \ \\:\_\ \ \     _\:\|    
  \:\ \ \ \\:\ \ \ \\:\\_  _\/ \::___\/_\: __ `\ \\:: __  \ \\::::_\/    /::_/__   
   \:\/.:| |\:\_\ \ \\:\_\ \ \  \:\____/\\ \ `\ \ \\:.\ \  \ \ \::\ \    \:\____/\ 
    \____/_/ \_____\/ \_____\/   \_____\/ \_\/ \_\/ \__\/\__\/  \__\/     \_____\/  )" << std::endl;

    std::cout << "V.2.0   by Philip Prager Urbina   2022" << std::endl;
    std::cout << "Find on github: https://github.com/PhilipPragerUrbina/DOGERAY2" << std::endl;
    //check what file to open. if none specified, try to open defualt file. Normally you use "open with" to open scenes.
    std::string filename = "defualt.gltf";
    if (argc > 1)  filename = args[1];
    //load file
    config settings;
    loader file(filename, &settings);
    file.loadGLTF();

    bvhtree tree(file.loadedtris);
    tree.build();
    int treesize = 0;
    bvhnode* finishedtree = tree.getNodes(treesize);


    //width and height
    int wi = 400;
    int h = 400;
    //create window and gui
    gui g("DOGEGUI",200,200);
    window win("PAIN",wi, h);
   
    //output data
    uint8_t* data; 

    //main loop
    int i = 0;
   
    settings.cam.position = vec3(-6, 0, 1);
    settings.cam.lookposition = vec3(0, 0, 0);
    settings.cam.calculate();
    settings.bvhsize = treesize;
    settings.h = h;
    settings.w = wi;
    tracekernel shader(settings, finishedtree);
    while (!g.exit) {
        //edit output data. Later will be moved to kernel
        data = win.getTex();
        shader.render(data, settings);
  

        //update 
        g.update(&settings);
        i++;
        win.update(data);

      
    }
    
    //clean up
    delete[] data;
  
  
  
    return 0;
}

